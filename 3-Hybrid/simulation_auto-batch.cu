#include "hip/hip_runtime.h"
/**
 * Programmation GPU 
 * Universite Pierre et Marie Curie
 * Simulation de transport de neutrons.
 *
 * M. Pecheux - Automne 2018
 * [MAIN5 - HPCA]
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

extern "C" double my_gettimeofday();

#define NB_BLOCKS  256
#define NB_THREADS 256

/** 
 * Controle des erreurs CUDA et debugging. 
 */

#ifdef CUDA_DEBUG
#define CUDA_SYNC_ERROR() {						\
    hipError_t sync_error;						\
    hipDeviceSynchronize();						\
    sync_error = hipGetLastError();					\
    if(sync_error != hipSuccess) {					\
      fprintf(stderr, "[CUDA SYNC ERROR at %s:%d -> %s]\n",		\
	      __FILE__ , __LINE__, hipGetErrorString(sync_error));	\
      exit(EXIT_FAILURE);						\
    }									\
  }
#else /* #ifdef CUDA_DEBUG */
#define CUDA_SYNC_ERROR()
#endif /* #ifdef CUDA_DEBUG */

#define CUDA_ERROR(cuda_call) {					\
    hipError_t error = cuda_call;				\
    if(error != hipSuccess){					\
      fprintf(stderr, "[CUDA ERROR at %s:%d -> %s]\n",		\
	      __FILE__ , __LINE__, hipGetErrorString(error));	\
      exit(EXIT_FAILURE);					\
    }								\
    CUDA_SYNC_ERROR();						\
  }

/*
 * Generates a random number for the given thread.
 */
 
__device__ float uniform_random_number(hiprandState* global_state, int thread_id) {
    hiprandState local_state = global_state[thread_id];
    float RANDOM            = hiprand_uniform(&local_state);
    global_state[thread_id] = local_state;
    return RANDOM;
}

__global__ void setup_kernel_seeds(hiprandState* state, unsigned long seed) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}

__global__ void kernel_simulation(float c, float c_c, float c_s, float h, int n,
                                  int* res, float* absorbed,
                                  hiprandState* global_state) {
  int i  = blockDim.x * blockIdx.x + threadIdx.x;
  int gi = i;
  int idx;
   
  float d; // direction du neutron (0 <= d <= PI)
  float x; // position de la particule (0 <= x <= h)
  float L; // distance parcourue par le neutron avant la collision
  float u; // variable aléatoire uniforme
  
  // memoire partagee entre les threads du bloc
  __shared__ int R[NB_THREADS];
  __shared__ int B[NB_THREADS];
  __shared__ int T[NB_THREADS];
  R[threadIdx.x] = 0;
  B[threadIdx.x] = 0;
  T[threadIdx.x] = 0;
  
  while(i < n) {
    d = 0.0; x = 0.0;
    while (1) {
      u = uniform_random_number(global_state, gi);
      L = -(1 / c) * log(u);
      x = x + L * cos(d);
      if (x < 0) {
      	R[threadIdx.x] = R[threadIdx.x] + 1; // ajout d'1 neutron reflechi
      	break;
      } else if (x >= h) {
      	T[threadIdx.x] = T[threadIdx.x] + 1; // ajout d'1 neutron transmis
      	break;
      } else if ((u = uniform_random_number(global_state, gi)) < c_c / c) {
      	B[threadIdx.x] = B[threadIdx.x] + 1; // ajout d'1 neutron absorbe
        idx = atomicAdd(res+3, 1); // recuperation + incrementation du compteur
                                   // de neutrons absorbes
      	absorbed[idx] = x;  // stockage de la position d'absorption
      	break;
      } else {
      	u = uniform_random_number(global_state, gi);
      	d = u * M_PI;
      }
    }
    i += gridDim.x * blockDim.x; // saut d'un bloc
  }

  /* REDUCTION */
  // synchronisation des threads du bloc
  __syncthreads();
  // calcul et mise a jour des compteurs globaux
  int j = blockDim.x / 2;
  while (j > 0) {
    if (threadIdx.x < j) {
      R[threadIdx.x] += R[threadIdx.x + j];
      B[threadIdx.x] += B[threadIdx.x + j];
      T[threadIdx.x] += T[threadIdx.x + j];
    }
    j /= 2;
    __syncthreads();
  }

  // ajout du bloc par le premier GPU du bloc
  if (threadIdx.x == 0) {
    atomicAdd(res,   R[0]);
    atomicAdd(res+1, B[0]);
    atomicAdd(res+2, T[0]);
  }
}

/**
 * Effectue la simulation de 'n' particules.
 */

extern "C"
void gpu_simulation(float c, float c_c, float c_s, float h, int n, int** res,
                    float** absorbed) {
  
  /* Variables liees au chronometrage */
  double debut, fin;

  /* GPU allocation */
  int s_res = 4 * sizeof(int);
  int s_abs = n * sizeof(float);
  int   *d_res;
  float *d_abs;
  CUDA_ERROR(hipMalloc((void**) &d_res, s_res));
  CUDA_ERROR(hipMalloc((void**) &d_abs, s_abs));

  /* CPU > GPU transfers (synchronous) */
  CUDA_ERROR(hipMemcpy(d_res, *res,      s_res, hipMemcpyHostToDevice));
  CUDA_ERROR(hipMemcpy(d_abs, *absorbed, s_abs, hipMemcpyHostToDevice));
  
  /* definition de la taille de la grille de GPUs */
  dim3 nbBlocks, nbThreads;
  nbThreads.x = NB_THREADS;
  nbThreads.y = nbThreads.z = 1;
  nbBlocks.x  = NB_BLOCKS;
  nbBlocks.y  = nbBlocks.z  = 1;

  printf("--------\n");
  printf("Nb blocs : %d\t\tNb threads par bloc : %d\n", nbBlocks.x, nbThreads.x);
  printf("[GPU] Taille de batch automatique.\n");

  /* variables pour la generation de nombres aleatoires */
  hiprandState* dev_states;
  CUDA_ERROR(hipMalloc((void**) &dev_states, nbThreads.x*nbBlocks.x*sizeof(hiprandState)));

  /* preparation des graines aleatoires */
  setup_kernel_seeds <<<nbBlocks, nbThreads>>>(dev_states, unsigned(time(NULL)));

  /* debut du chronometrage */
  debut = my_gettimeofday();            

  /* lancement des kernels */
  kernel_simulation<<<nbBlocks, nbThreads>>>(c, c_c, c_s, h, n, d_res, d_abs, dev_states);

  /* GPU > CPU transfers (synchronous) */
  CUDA_ERROR(hipMemcpy(*res,      d_res, s_res, hipMemcpyDeviceToHost));
  CUDA_ERROR(hipMemcpy(*absorbed, d_abs, s_abs, hipMemcpyDeviceToHost));

  /* fin du chronometrage */
  fin = my_gettimeofday();
  printf("[GPU] Temps de calcul seul : %.10f seconde(s)\n", fin - debut);

  /* liberation memoire */
  CUDA_ERROR(hipFree(d_res));
  CUDA_ERROR(hipFree(d_abs));
  CUDA_ERROR(hipFree(dev_states));
}
